#include<stdio.h>
#include<assert.h>
#include<math.h>
#include<hip/hip_runtime.h>
#define N 1000000

#define HANDLE_ERROR( err )(handleCudaError( err, __FILE__, __LINE__ ) )

int handleCudaError(hipError_t cut,const char* file, int line)
{
	if(cut != hipSuccess)
		{
		printf("%s : File: %s  Line: %d \n",hipGetErrorString(cut),file,line);
		return -1 ;
		}
	return 0;  
}


//================== CUDA FUNCTIONS ====================

__global__ void random_init(int n, int *x, int*y)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread
	
	if (i < n)
	{
	x[i] = ((unsigned long long int)950706376*i) % 0x7FFFFFFFlu;
	y[i] = ((unsigned long long int)950706376*i) % 0x7FFFFFFFlu;
	}
	
}




// calculate points that are in the circle 

__global__ void calculate_n(int n, int *x, int*y, int *num_points)
{

int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread

if(i < n)
	{
		if(sqrt((float)(x[i]*x[i]) +(float)(y[i]*y[i])) <= 0x7FFFFFFFlu)
			{
				atomicAdd(num_points,1);	
			}
	}

}


int main(int argc, char* argv[])
{
	// ============= INIT =====================

	int sum = 0; 
	int *random_points_x_d = NULL;
	int *random_points_y_d = NULL;
	int *num_points_d = NULL;
	float pi =0; 

	 	

	//============TRANSFER======================
	HANDLE_ERROR(hipMalloc(&random_points_x_d, sizeof(int)*N)); // malloc of x_device
	HANDLE_ERROR(hipMalloc(&random_points_y_d , sizeof(int)*N)); // malloc of y_device

	HANDLE_ERROR(hipMalloc(&num_points_d, sizeof(int))); //malloc of n_points
	HANDLE_ERROR(hipMemcpy(num_points_d, &sum, sizeof(int), hipMemcpyHostToDevice));

	
	//=============Calculation ==================
	

	random_init<<<ceil((float)N/(float)256),256>>>(N,random_points_x_d, random_points_y_d);
	calculate_n<<<ceil((float)N/(float)256),256>>>(N,random_points_x_d, random_points_y_d, num_points_d);




	//===========CHECK============================
 
	
	HANDLE_ERROR(hipMemcpy(&sum,num_points_d, sizeof(int), hipMemcpyDeviceToHost));
	pi = 4 * (sum / N) ;


	printf("%f",pi);


//=========CLEAN==============================

	HANDLE_ERROR(hipFree(random_points_x_d));
	HANDLE_ERROR(hipFree(random_points_y_d));
	HANDLE_ERROR(hipFree(num_points_d));

	return 0 ; 

}

