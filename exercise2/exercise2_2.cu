#include<stdio.h>
#include<assert.h>
#include<hip/hip_runtime.h>
#define N 1000000

#define HANDLE_ERROR( err )(handleCudaError( err, __FILE__, __LINE__ ) )

int handleCudaError(hipError_t cut,const char* file, int line)
{
	if(cut != hipSuccess)
		{
		printf("%s : File: %s  Line: %d \n",hipGetErrorString(cut),file,line);
		return -1 ;
		}
	return 0;  
}


__global__ void parallel_add(int n, int *a ,int *b , int *c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread
	if(i < n)
	{
		c[i] = a[i] + b[i]; // simple add 
	}
}

__global__ void parallel_scalar_product(int n, int *a,int *b, int *erg)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread
	if(i < n)
	{	
		int product = a[i] * b[i];
		atomicAdd(erg,product); // simple add 
	}


}

int main(int argc, char* argv[])
{
	// ============= INIT =====================
	int *a_host= NULL;
	int *b_host=NULL;
	int erg_host=0;
	int *a_device_ptr = NULL ;
	int *b_device_ptr = NULL ;
	int *erg_device_ptr = NULL ; 
	 	
	

	a_host=(int*)malloc(sizeof(int)*N);
	b_host=(int*)malloc(sizeof(int)*N);


	
	for(unsigned int i = 0; i < N; ++i)
	{
		a_host[i] = 1 ;
		b_host[i] = 1;
	}


	//============TRANSFER======================
	HANDLE_ERROR(hipMalloc(&a_device_ptr, sizeof(int)*N)); // malloc of a_device
	HANDLE_ERROR(hipMalloc(&b_device_ptr, sizeof(int)*N)); // malloc of b_device
	HANDLE_ERROR(hipMalloc(&erg_device_ptr, sizeof(int))); // malloc of erg_device 
	
	//Transfer a_host to a_device
	HANDLE_ERROR(hipMemcpy(a_device_ptr, a_host, sizeof(int)*N, hipMemcpyHostToDevice));
		
	//Transfer b_host to b_device
	HANDLE_ERROR(hipMemcpy(b_device_ptr, b_host, sizeof(int)*N, hipMemcpyHostToDevice));
	
	HANDLE_ERROR(hipMemcpy(erg_device_ptr, &erg_host, sizeof(int), hipMemcpyHostToDevice));

	
	//=============Calculation ==================
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);	

	hipEventRecord(start);
	parallel_scalar_product<<<ceil((float)N/(float)256),256>>>(N,a_device_ptr,b_device_ptr,erg_device_ptr);
	hipEventRecord(stop);



	//===========CHECK============================
 
	
	HANDLE_ERROR(hipMemcpy(&erg_host,erg_device_ptr, sizeof(int), hipMemcpyDeviceToHost));
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time %f milliseconds \n", milliseconds) ; 

/*	int erg = 0
	for(unsigned int i = 0; i < N; ++i)
	{
		//correct_transfer = correct_transfer & (a_host[i] == b_host[i]); 	
		erg += a[i] *b[i] ;

	} */

	if(erg_host == N)
	{
	printf("Correct Calculation \n");
	
	}
	else
	{
	printf(" Non Correct Calculation %d %d \n", erg_host , N);
	}
 	//============CLEAN==============================

	HANDLE_ERROR(hipFree(a_device_ptr));
	HANDLE_ERROR(hipFree(b_device_ptr));
	HANDLE_ERROR(hipFree(erg_device_ptr));
	free(a_host);
	free(b_host);
	
	a_host= NULL;
	b_host= NULL; 

	return 0 ; 

}

