#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>


#define HANDLE_ERROR( err )(handleCudaError( err, __FILE__, __LINE__ ) )





__global__ void diadic_Product (int n, int *a,int *b, int *erg)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread in x
	int j = blockIdx.y * blockDim.y + threadIdx.y; // Calculate current Tread in y
	
	if(i < n && j < n)
	{	
		
		erg[i*n+j] = a[i]*b[j]; 	
	
	}
}

__global__ void matrixProduct(int n, int *a, int *b, int *c)
{
		int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate current Thread in x
		int j = blockIdx.y * blockDim.y + threadIdx.y; // Calculate current Tread in y
		
		int scalarProduct = 0; 
		for(int k = 0; k < n; ++k)
		{
			scalarProduct += a[i*n+k]* b[k*n+j];
		}
		c[i*n+j] = scalarProduct ;
	
	
}

int handleCudaError(hipError_t cut,const char* file, int line)
{
	if(cut != hipSuccess)
		{
		printf("%s : File: %s  Line: %d \n",hipGetErrorString(cut),file,line);
		return -1 ;
		}
	return 0;  
}

void matrixProduct(int n, int* a, int* b ,int *erg)
{
	for (int i = 0 ; i < n ; ++i)
	{
		
		for(int j = 0 ; j < n; ++j)
		{
			int scalarProduct = 0; 
			for(int k = 0 ; k < n ; ++k)
			{
				//sclarproduct of i'th row and j'th collumn 
				int scalarProduct += a[i*n+k]* b[k*n+j] ;
			}
			
			erg[i*n+j]= scalarProduct;
		}
		
	}
	
	
}



void diadicProduct(int n, int *a , int *b, int *erg)   // erg = a * b
{
	for(int i=0; i < n; ++i)
		{
			
			for ( int j = 0; j < n; ++j)
				{
					
					erg[i*n+j] = a[i]*b[j]; 
					
				}
		
		}
	
	

}

void printVector(int *vector, int n)
{
	for(int i = 0 ; i < n; ++i)
		{
			printf("  %d \n", vector[i]);	
					
		}
}

void printMatrix(int *matrix,int n)
{
		// print matrix a 
	for(int i = 0; i < n; ++i)
		{
		for (int j = 0 ; j < n; ++j)
			{
				printf("%d",matrix[n*i+j]);
			}	
			printf("\n");
		}
 }






int main(int argc, char**args)
{
	if(argc != 2) 
		{
			printf("Call Programm with program ./programm.out <dimension>");
			return -1 ;
		}
	
	int n = atoi(args[1]) ;
	time_t t;
	int *a = (int *) malloc( sizeof(int)*n*n);
	int *b = (int *) malloc( sizeof(int)*n*n);
	int *c = (int *) malloc( sizeof(int)*n*n);
	int *c_t = (int *) malloc( sizeof(int)*n*n);

	int *a_d = NULL ;
	int *b_d = NULL ;
	int *c_d = NULL ;


	HANDLE_ERROR(hipMalloc(&a_d, sizeof(int)*n)); // malloc of a_device
	HANDLE_ERROR(hipMalloc(&b_d, sizeof(int)*n)); // malloc of b_device
	HANDLE_ERROR(hipMalloc(&c_d, sizeof(int)*n*n)); // malloc of c_device 
	double time1=0.0, tstart;      // time measurment variables
	// random init 
	time(&t);
	srand((unsigned int)t);   


	for(int i = 0; i < n; ++i)
		{
			for (int j= 0; j < n ; ++j)
			{
					a[i*n+j] = rand() % 5 ;	
					b[i*n+j]= rand() % 5 ; 		
			}	
		
		}

	//Transfer a_host to a_device
	HANDLE_ERROR(hipMemcpy(a_d, a, sizeof(int)*n*n, hipMemcpyHostToDevice));
		
	//Transfer b_host to b_device
	HANDLE_ERROR(hipMemcpy(b_d, b, sizeof(int)*n, hipMemcpyHostToDevice));
	

	

	printf("=============MATRIX A =============  \n");
	printVector(a,n);



	printf("===============MATRIX B================== \n");
	printVector(b,n);

	
	
	printf("====== Result of matrix multiplication =====\n");
	tstart = clock();              // start 
	diadicProduct(n,a,b,c);
	time1 = clock() - tstart;     // end
	time1 = time1/CLOCKS_PER_SEC;  // rescale to seconds
	

	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	dim3 block(16,16,1);
	dim3 grid(ceil((float)n/(float)16),ceil((float)n/(float)16),1);
	diadic_Product<<<grid,block>>>(n,a_d,b_d,c_d);
	HANDLE_ERROR(hipEventRecord(stop, 0));
	HANDLE_ERROR(hipEventSynchronize(stop));
	HANDLE_ERROR(hipMemcpy(c_t, c_d, sizeof(int)*n*n, hipMemcpyDeviceToHost));
	printf("====== Result of matrix multiplication on Kernel=====\n");
	

	for(int i = 0; i < (n*n); ++i)
		{
			if(c[i] != c_t[i])
				{
					printf("failure at %d",i);
					break; 
				}
		}

	HANDLE_ERROR(hipFree(a_d));
	HANDLE_ERROR(hipFree(b_d));
	HANDLE_ERROR(hipFree(c_d));
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel-diadicProduct: %f msec\n", time );
	printf ("Time for the CPU -diadicProduct %d msec \n",time1 *1000);

	return 0;  	

}
